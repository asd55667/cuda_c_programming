#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../common/common_win.h"

int main()
{
    int dev = 0;
    CHECK(hipSetDevice(dev));
    
    unsigned int size = 1<<24;
    unsigned int nbytes = size * sizeof(float);
    hipDeviceProp_t devp;
    CHECK(hipGetDeviceProperties(&devp, dev));
    printf("Dev: %d, name %s, size %5.2fMB\n", dev, devp.name, nbytes/1024.f/1024.f);

    if(!devp.canMapHostMemory)
    {
        printf("Device %d doesn't support mapping HOST mem\n", dev);
        CHECK(hipDeviceReset());
        return EXIT_SUCCESS;
    }

    float *h;
    CHECK(hipHostMalloc((void **)&h, nbytes));
    memset(h, 0, nbytes);
    init_float(h, size);

    float *d;
    CHECK(hipMalloc((void **) &d, nbytes));

    CHECK(hipMemcpy(d, h, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(h, d, nbytes, hipMemcpyDeviceToHost));

    CHECK(hipHostFree(h));
    CHECK(hipFree(d));

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}