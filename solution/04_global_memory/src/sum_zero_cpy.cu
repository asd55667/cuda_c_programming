#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include "../common/common_win.h"

void cpu_sum(float *a, float *b, float *c, const int n)
{
    for (int i=0; i< n; i++)
    {
        c[i] = a[i] + b[i];
    }
}

__global__ void gpu_sum(float *a, float *b, float *c, unsigned int n)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

__global__ void gpu_sum_zero_cpy(float *a, float *b, float *c, unsigned int n)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}


int main(int argc, char **argv)
{
    int dev = 0;
    hipDeviceProp_t devp;
    CHECK(hipSetDevice(dev));
    CHECK(hipGetDeviceProperties(&devp, dev));
    printf("Dev %d, name %s\n", dev, devp.name);

    if (!devp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        CHECK(hipDeviceReset());
        exit(EXIT_SUCCESS);
    }

    int pow = 22;
    // if (argc > 1) pow = atoi(argv[1]);
        
    unsigned int size = 1 << pow;
    unsigned int nbytes = size * sizeof(float);
    printf("Vec size %5.2fMB\n", nbytes/1024.f/1024.f);

    // cpu alloc
    float *ha, *hb, *hc, *hd;
    ha = (float *)malloc(nbytes);
    hb = (float *)malloc(nbytes);
    hc = (float *)malloc(nbytes);
    hd = (float *)malloc(nbytes);

    init_float(ha, size);
    init_float(hb, size);
    memset(hc, 0, nbytes);
    memset(hd, 0, nbytes);

    float *da, *db, *dc;
    CHECK(hipMalloc((void **)&da, nbytes));
    CHECK(hipMalloc((void **)&db, nbytes));
    CHECK(hipMalloc((void **)&dc, nbytes));
    
    CHECK(hipMemcpy(da, ha, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(db, hb, nbytes, hipMemcpyHostToDevice));
    
    int nthread = 512;
    dim3 block(nthread);
    dim3 grid((size+block.x-1)/block.x);
    printf("Cfg <<<%d,%d>>>\n",grid.x, block.x);
    
    cpu_sum(ha, hb, hc, size);
    gpu_sum<<<grid, block>>>(da, db, dc, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    
    CHECK(hipMemcpy(hd, dc, nbytes, hipMemcpyDeviceToHost));
    check_float(hd, hc, size);

    CHECK(hipFree(da));
    CHECK(hipFree(db));
    free(ha);
    free(hb);

    // zero-cpy
    printf("---------zero-copy mem------------")
    CHECK(hipHostAlloc((void **) &ha, nbytes, hipHostMallocMapped));
    CHECK(hipHostAlloc((void **) &hb, nbytes, hipHostMallocMapped));
    init_float(ha, size);
    init_float(hb, size);
    memset(hc, 0, nbytes);
    memset(hd, 0, nbytes);

    CHECK(hipHostGetDevicePointer((void **)&da, (void *)ha, 0));
    CHECK(hipHostGetDevicePointer((void **)&db, (void *)hb, 0));
    
    
    cpu_sum(ha, hb, hc, size);
    gpu_sum_zero_cpy<<<grid, block>>>(da, db, dc, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(hd, dc, nbytes, hipMemcpyDeviceToHost))   ;
    //printf("Result of GPU sum zero-copy %f\n",hd);
    
    check_float(hc, hd, size);

    CHECK(hipHostFree(da));
    CHECK(hipHostFree(db));
    CHECK(hipFree(dc));
    free(hc);
    free(hd);

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}