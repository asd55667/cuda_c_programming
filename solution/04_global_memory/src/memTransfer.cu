#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../common/common_win.h"

int main()
{
    int dev = 0;
    CHECK(hipSetDevice(dev));

    unsigned int size = 1 << 24;
    unsigned int nbytes = size * sizeof(float);
    hipDeviceProp_t devp;
    CHECK(hipGetDeviceProperties(&devp, dev));
    printf("Dev: %d, name %s, size %5.2fMB\n",dev, devp.name, nbytes/1024.f/1024.f);

    float *h = (float *)malloc(nbytes);
    init_float(h, size);

    float *d;
    CHECK(hipMalloc((void **)&d, nbytes));

    CHECK(hipMemcpy(d, h, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(h, d, nbytes, hipMemcpyDeviceToHost));


    CHECK(hipFree(d));
    free(h);

    

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
