#include<hip/hip_runtime.h>
#include<stdio.h>

int main()
{
    int num = 0;
    int maxdev = 0;
    hipGetDeviceCount(&num);
    if (num > 1)
    {
        int maxp = 0;
        for (int i = 0; i < num; i++)
        {
            hipDeviceProp_t props;
            hipGetDeviceProperties(&props, i);
            if (maxp < props.multiProcessorCount)
            {
                maxp = props.multiProcessorCount;
                maxdev = i;
            }
        }
        hipSetDevice(maxdev);
    }
    printf("dev idx: %d\n", maxdev);
}
